#include "hip/hip_runtime.h"
#include "montecarlo.hpp"

//TODO deflectionAngleArray accesso Random blocca
//TODO float to float o float16

__constant__  float deflectionAngleArray_const[SIZE_LIST_ANGLE];

__host__ Results::Results()
    {
    for(std::size_t i = 0; i<this->tcspc.size(); ++i)
        this->tcspc[i] = 0;
    
    for(std::size_t i = 0; i<this->cos_angle.size(); ++i)
        this->cos_angle[i] = 0;
    }
__host__ float henyey_greenstein_F(const float &theta, const float &g)
    {
    if(g == 0)
        return (1-cos(theta))/2;
    // Calculate the CDF for the Henyey-Greenstein phase function - Martelli's book 9.12 old version
    return (1-g*g)/(2*g)*(1/(1-g) -1/sqrt(1+g*g-2*g*cos(theta)));
    }
__host__ std::array<float, SIZE_LIST_ANGLE> inverse_transform_sampling(std::function<float( const float &, const float &)> cdf, const float &g)
    {
    std::array<float, SIZE_LIST_ANGLE> deflectionAngleArray;
    for(std::size_t i = 0; i<SIZE_LIST_ANGLE; ++i)
        {
        float x = 0;    // initial value for x
        float cdf_val = cdf(x, g);    // initial value for CDF(x)

        // Find the value x that satisfies F(x) = u
        while (cdf_val < float(i)/SIZE_LIST_ANGLE)
            {
            x += 1e-4;    // increment x
            cdf_val = cdf(x, g);    // update CDF(x)
            }
       deflectionAngleArray[i] = x;
        }
    return deflectionAngleArray;
    }



__device__ int sign(const float &x)
    {
    if(x>0)
        return 1;
    return -1;
    }
__host__ __device__ Vector::Vector(const float &x_,const float &y_, const float &z_):
    x{x_},y{y_},z{z_}
    {}
__device__ Vector::Vector(const Vector &v):
    x{v.x},y{v.y},z{v.z}
    {}
__device__ Vector Vector::operator *(const float &a)
    {
    Vector v(a*this->x, a*this->y, a*this->z);
    return v;
    }
__device__ float Vector::operator *(const Vector &a)
    {
    return a.x*this->x + a.y*this->y + a.z*this->z;
    }
__device__ Vector Vector::operator +(const Vector &a)
    {
    Vector v(a.x+this->x, a.y+this->y, a.z+this->z);
    return v;
    }
__device__ Vector& Vector::operator +=(const Vector &a)
    {
    this->x += a.x;
    this->y += a.y;
    this->z += a.z;
    return *this;
    }


__device__ Photon::Photon(const Vector &position_,const Vector &direction_, const float &mu_s_):
    position(position_),
    direction(direction_),
    mu_s(mu_s_)
    {
    
    length = 0;
    time = 0;
    }


__host__ __device__ Detector::Detector(const Vector &position_, const float radius_):
    position(position_),
    radius{radius_}{}
__device__ bool Detector::is_recorded(const Photon &photon, const Vector &previous_position)
    {
    if (previous_position.z < 0 && photon.position.z>0){
        float t =  -photon.position.z/photon.direction.z;
        float x_intersection = photon.direction.x*t+ photon.position.x;
        float y_intersection = photon.direction.y*t+ photon.position.y;
        float x = x_intersection - this->position.x;
        float y = y_intersection - this->position.y;
        //std::cout<<x_intersection << " "<<x_det<<" "<<y_intersection << " "<<y_det<<std::endl;
        if (x*x+y*y< this->radius*this->radius)
            return true;
        }
    return false;
    }




__device__ void find_v1(Vector &v1, const Vector &v0, float theta, float phi)
    {
    if(fabs(v0.z)>0.99)
        {
        v1.x= sin(theta)*cos(phi);
        v1.y = sin(theta)*sin(phi);
        v1.z = sign(v0.z)*cos(theta);
        return;
        }
    float c0 = sin(theta)/sqrt(1.-v0.z*v0.z);    //preso da mcx_core.cu riga 931
    v1.x = c0*(v0.x*v0.z*cos(phi)-v0.y*sin(phi))+v0.x*cos(theta);
    v1.y = c0*(v0.y*v0.z*cos(phi)+v0.x*sin(phi))+v0.y*cos(theta);
    v1.z =  -c0*(1.-v0.z*v0.z)*cos(phi)+v0.z*cos(theta);
    //std::cout<< v1*v0<<" "<<cos(theta)<<std::endl;
    }
__device__ void Photon::generateRandomDirection(hiprandState_t &state)
    {// generate a random direction of scattering
    // Generate the scattering angle using the inverse CDF of the provided probability function
    float theta = deflectionAngleArray_const[int(hiprand_uniform(&state)*(SIZE_LIST_ANGLE-1))];
    //std::cout<<cos(theta)<<" ";
    float phi = 2 * PI * hiprand_uniform(&state);
    Vector u(this->direction);
    find_v1(this->direction, u, theta, phi);

    }
__device__ void Photon::propagatePhoton(hiprandState_t &state)
    {
    float dl = -log(1-hiprand_uniform(&state))/this->mu_s;//TODO check log
    //float dt = dl/C_LIGHT;
    this->length+=dl;
    generateRandomDirection(state);
    this->position += this->direction*dl;    // Propagate the photon in new direction
    this->time = length/C_LIGHT; // in ns
    }

__global__ void propagation(float mu_s, float g, int *tcspc, Detector *detector, int NUM_PHOTONS, int PHOTON_INTEGRATION)
    {//TODO come inizilizzare una classe Detector
    int tot = 0;
    int repetition_per_thread = NUM_PHOTONS/(blockDim.x*gridDim.x);
    bool finish = false;
    hiprandState_t state;
    hiprand_init(blockIdx.x * blockDim.x + threadIdx.x, 0, 0, &state);
    for( int i = 0; i<repetition_per_thread && !finish; ++i) //in kernel
        {
        Vector position_start(0,0,0);
        Vector direction_start(0,0,-1);
        Photon photon(position_start, direction_start, mu_s);
        // Propagate the photon through the medium
        while (photon.time < TIME_LIMIT) 
            {
            Vector position_previous(photon.position);// Store the current position in a temporary variable
            Vector direction_previous(photon.direction);
            photon.propagatePhoton(state);
            Vector direction_new(photon.direction);
            if (detector->is_recorded(photon, position_previous))
                {
                if(int(photon.time*CH_PER_UNIT)>TIME_LIMIT*CH_PER_UNIT) break;
                atomicAdd(&tcspc[int(photon.time*CH_PER_UNIT)],1);
                ++tot;
                if(tot > PHOTON_INTEGRATION/(blockDim.x*gridDim.x)) finish = true;//return tcspc;  
                break;
                }  
            }
        }
    
    }

Results simulate(const float &g, const float &mu_s, Detector &detector, int NUM_PHOTONS, int PHOTON_INTEGRATION)
    {
    std::array<float, SIZE_LIST_ANGLE> deflectionAngleArray;
    // Create a random number generator
    deflectionAngleArray = inverse_transform_sampling(henyey_greenstein_F, g);

    Results res;
    int *tcspc;
    
    hipEvent_t start, stop;
    check(hipEventCreate(&start));
    check(hipEventCreate(&stop));

    Detector *detector_dvc;
    check(hipMalloc(&detector_dvc, 1*sizeof(Detector)));
    check(hipMemcpy(detector_dvc,&detector, 1*sizeof(Detector), hipMemcpyHostToDevice));
    check(hipMalloc(&tcspc, res.tcspc.size()*sizeof(int)));
    check(hipMemset(tcspc,0, res.tcspc.size()*sizeof(int)));
    check(hipMemcpyToSymbol(HIP_SYMBOL(deflectionAngleArray_const),deflectionAngleArray.data(), SIZE_LIST_ANGLE*sizeof(float)));
    dim3 dimBlock(1024);
    dim3 dimThreads(32);//TODO come settare i numeri??
    check(hipEventRecord(start, 0));
    propagation<<<dimBlock,dimThreads>>>(mu_s, g, tcspc, detector_dvc, NUM_PHOTONS, PHOTON_INTEGRATION);
    
    hipDeviceSynchronize();
    CHECK_LAST_CUDA_ERROR()
    check(hipEventRecord(stop, 0));
    float time;
    check(hipEventElapsedTime(&time, start, stop));
    printf("elapsed Time %f:", time);
    check(hipEventDestroy(start));
    check(hipEventDestroy(stop));
    
    check(hipMemcpy(res.tcspc.data(),tcspc, res.tcspc.size()*sizeof(int), hipMemcpyDeviceToHost));
    check(hipFree(tcspc));
    check(hipFree(detector_dvc));

    
    return res;
    }
int main(int argc, char* argv[])
    {
    if (argc < 3) 
        {
        std::cerr << "Usage: " << argv[0] << " <NUM_PHOTONS> <PHOTON_INTEGRATION>" << std::endl;
        return 1;
        }

    const int NUM_PHOTONS = std::atoi(argv[1]);
    const int PHOTON_INTEGRATION = std::atoi(argv[2]);
    Vector pos(1,0,0);//è una funzione __device__
    Detector d(pos,0.1);
    simulate(0, 10, d, NUM_PHOTONS, PHOTON_INTEGRATION);
    return 0;
    }
